#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int a, int b, int* c)
{
	*c = a+b;
}

int main(void)
{
	int h_c;
	int * d_c;
	
	hipMalloc((void**) &d_c, sizeof(int));
	
	add<<<1,1>>>(2,7,d_c);
	
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("2+7 = %d\n",h_c);
	
	hipFree(dev_c);
}
