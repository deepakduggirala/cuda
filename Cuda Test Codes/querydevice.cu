#include<stdio.h>

int main(void)
{
	int count;
	
	hipDeviceProp_t prop;
	
	hipGetDeviceCount(&count);
	printf("No:of devices: %d\n\n",count);
	
	for(int i=0; i<count; i++)
	{
		hipGetDeviceProperties( &prop, i);
		printf( "--- General Information for device %d ---\n", i );
		printf( "Name:%s\n", prop.name );
		printf( "Compute capability:%d.%d\n", prop.major, prop.minor );
		printf( "Clock rate:%d\n", prop.clockRate );
		printf( "Max Threads per block: %d\n",prop.maxThreadsPerBlock );
	}
}
