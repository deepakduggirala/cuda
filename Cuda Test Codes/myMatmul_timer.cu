#include "hip/hip_runtime.h"
#include <stdio.h>
#include<time.h>
#include<stdlib.h>
using namespace std;

#define M 2
#define N 2
#define K 2

__global__ void matMulKernel(float* A, float* B, float* C)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= M || col >= N) return;
	float sum =0;
	for(int i=0;i<N;i++)
	{
		sum += A[row*N + i]*B[i*K + col];
	}
	C[row*K + col] = sum;
}

timespec diff(timespec start, timespec end)
{
	timespec temp;
	if ((end.tv_nsec-start.tv_nsec)<0) {
		temp.tv_sec = end.tv_sec-start.tv_sec-1;
		temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec-start.tv_sec;
		temp.tv_nsec = end.tv_nsec-start.tv_nsec;
	}
	return temp;
}

int main(void)
{	
	
	//decalre host matrices
	float hA[M*N],hB[N*K],hC[M*K];
	
	timespec time1, time2, time3;
	
	//populate host matrices
	for(int i=0;i<M;i++)
	{
		for(int j=0;j<N;j++)
		{
			hA[i*N+j] = (float)(rand()%10);
//			printf("%f ",hA[i*N+j]);
		}
//		printf("\n");
	}
//	printf("\n");
	
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<K;j++)
		{
			hB[i*K+j] = (float)(rand()%10);
//			printf("%f ",hB[i*K+j]);
		}
//		printf("\n");
	}
//	printf("\n");
	
	

	//load dA and dB into device memory
	float* dA, *dB, *dC;
	hipError_t err;
	uint size = M*N*sizeof(float);
	err = hipMalloc((void**)&dA, size);
	printf("CUDA malloc A: %s\n",hipGetErrorString(err));
	err = hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n",hipGetErrorString(err)); 
	
	
	size = N*K*sizeof(float);
	err = hipMalloc((void**)&dB, size);
	printf("CUDA malloc B: %s\n",hipGetErrorString(err));
	err = hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n",hipGetErrorString(err));
	
	//create dC in device memory
	size = N*K*sizeof(float);
	err = hipMalloc((void**)&dC, size);
	printf("CUDA malloc C: %s\n",hipGetErrorString(err));
	
	//kernel
	
	dim3 blockSize(16,32);
	dim3 gridSize;
	gridSize.x = (K+15)/16;
	gridSize.y = (M+31)/32;

	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

	matMulKernel<<<gridSize, blockSize>>>(dA, dB, dC);
	err = hipDeviceSynchronize();
  	printf("Run kernel: %s\n", hipGetErrorString(err));
  	
  	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
  	
  	// Read C from device memory
  	err = hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost); 
  	printf("Copy C off of device: %s\n",hipGetErrorString(err));
	
	//Free device memory
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	
	
	
	time3 = diff(time1,time2);
	printf("%ld:%ld\n",time3.tv_sec, time3.tv_nsec);
	
/*	for(int i=0;i<M;i++)
	{
		for(int j=0;j<K;j++)
		{
			printf("%f ",hC[i*K+j]);
		}
		printf("\n");
	}*/
}

